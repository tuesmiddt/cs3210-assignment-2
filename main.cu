#include "hip/hip_runtime.h"
#include "hash.h"
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

/* Magical endian switching macros */
#define BSWAP32(val) (((((val) >> 24) & 0xFF)) | ((((val) >> 16) & 0xFF) << 8) | ((((val) >> 8) & 0xFF) << 16) | (((val)&0xFF) << 24))
#define BSWAP64(val) (((((val) >> 56) & 0xFF)) | ((((val) >> 48) & 0xFF) << 8) | ((((val) >> 40) & 0xFF) << 16) | ((((val) >> 32) & 0xFF) << 24) | ((((val) >> 24) & 0xFF) << 32) | ((((val) >> 16) & 0xFF) << 40) | ((((val) >> 8) & 0xFF) << 48) | ((((val)) & 0xFF) << 56))

// #define DEBUG
// #define VERBOSE

// Tesla V100: 84 SMs, each with 64 INT32 cores
// #define NUM_BLOCKS 84
// #define NUM_THREADS 64
// #define OFFSET_STEP ((NUM_BLOCKS) * (NUM_THREADS))

/* Size of each field in input in bytes */
#define TIME_SIZE 4
#define DIGEST_SIZE 32
#define NUSNET_ID_SIZE 8
#define NONCE_SIZE 8

/* STATES */
__managed__ uint64_t nonce_found;
__managed__ uint8_t result[DIGEST_SIZE];
__managed__ int is_found = 0;

/* FOR CONSTRUCTING INPUT */
#define NUSNET_ID "E0014691"
__constant__ uint32_t cur_time_be;
__constant__ uint8_t digest[DIGEST_SIZE];
__constant__ char nusnet_id[NUSNET_ID_SIZE + 1] = NUSNET_ID;
__constant__ uint64_t target; // target value is 64 bit

__device__ void construct_input(uint8_t __restrict__ input[52], uint64_t nonce)
{
    size_t cur = 0;
    uint64_t nonce_be = BSWAP64(nonce);

    // Fill in the timestamp
    memcpy(input, &cur_time_be, TIME_SIZE);
    cur += TIME_SIZE;
    // Fill in previous digest
    memcpy((uint8_t*)input + cur, digest, DIGEST_SIZE);
    cur += DIGEST_SIZE;
    // Fill in NUSNET ID
    memcpy((uint8_t*)input + cur, nusnet_id, NUSNET_ID_SIZE);
    cur += NUSNET_ID_SIZE;
    // Fill in nonce
    memcpy((uint8_t*)input + cur, &nonce_be, NONCE_SIZE);
}

__global__ void find_hash(uint64_t offset, int num_threads)
{
    int prev_is_found;
    uint8_t input[52], hash[32];
    uint64_t to_compare;
    uint64_t nonce = offset + blockIdx.x * num_threads + threadIdx.x;
    if (is_found) {
        return;
    }
    construct_input(input, nonce);
    sha256(hash, input, 52);
    memcpy(&to_compare, hash, sizeof(uint64_t));
    if (to_compare < target) {
        // Test-and-set to prevent race condition of multiple writes
        prev_is_found = atomicExch(&is_found, 1);
        if (!prev_is_found) {
#ifdef DEBUG
            for (int i = 0; i < 52; i++) {
                printf("%x, ", input[i]);
            }
            printf("\n");
#endif
            nonce_found = nonce;
            memcpy(result, hash, DIGEST_SIZE);
        }
    }
}

void process_digest(char prev_hash[65], uint8_t digest[DIGEST_SIZE])
{
    int i;
    char cur_byte_string[3];
    // Set last element to null
    cur_byte_string[2] = 0;
    for (i = 0; i < DIGEST_SIZE; i++) {
        cur_byte_string[0] = prev_hash[i * 2];
        cur_byte_string[1] = prev_hash[(i * 2) + 1];
        digest[i] = strtol(cur_byte_string, NULL, 16);
    }
}

void print_digest(uint8_t __restrict__ digest[DIGEST_SIZE])
{
    int i;
    for (i = 0; i < DIGEST_SIZE; i++) {
        printf("%02x", digest[i]);
    }
    printf("\n");
}

int main(int argc, char** argv)
{
    int num_blocks, num_threads;
    char prev_hash[65]; // SHA-256 is 64 chars long
    uint64_t target_local;
    uint8_t digest_local[DIGEST_SIZE];
    uint32_t cur_time_local = time(NULL);
    uint32_t cur_time_local_be = BSWAP32(cur_time_local);
    uint64_t offset = 0;

    if (argc != 3) {
        printf("Usage:\n%s num_blocks num_threads\n", argv[0]);
        return 1;
    }

    num_blocks = atoi(argv[1]);
    num_threads = atoi(argv[2]);

    scanf("%s", prev_hash);
    scanf("%lu", &target_local);
    process_digest(prev_hash, digest_local);

    hipMemcpyToSymbol(HIP_SYMBOL(cur_time_be), &cur_time_local_be, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(target), &target_local, sizeof(uint64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(digest), &digest_local, DIGEST_SIZE * sizeof(uint8_t));

#ifdef DEBUG
    puts("Input:");
    printf("%s\n", prev_hash);
    printf("%lu\n", target_local);
#endif

    while (!is_found) {
#ifdef VERBOSE
        printf("Trying offset %lu\n", offset);
#endif
        find_hash<<<num_blocks, num_threads>>>(offset, num_threads);
        offset += num_blocks * num_threads;
    }
    hipDeviceSynchronize();

    puts(NUSNET_ID);
    printf("%u\n", cur_time_local);
    printf("%lu\n", nonce_found);
    print_digest(result);
    return 0;
}
