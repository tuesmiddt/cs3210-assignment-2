#include "hip/hip_runtime.h"
#include "hash.h"
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

// #define SEQUENTIAL
// #define DEBUG

// Tesla V100: 84 SMs, each with 64 INT32 cores
#define NUM_BLOCKS 84
#define NUM_THREADS 64
#define OFFSET_STEP ((NUM_BLOCKS) * (NUM_THREADS))

/* Size of each field in input in bytes */
#define TIME_SIZE 4
#define DIGEST_SIZE 32
#define NUSNET_ID_SIZE 8
#define NONCE_SIZE 8

/* STATES */
__managed__ uint64_t offset = 0;
__managed__ uint64_t nonce_found;
__managed__ uint8_t result[DIGEST_SIZE];
__managed__ int is_found = 0;

/* FOR CONSTRUCTING INPUT */
#define NUSNET_ID "E0014691"
__constant__ uint32_t cur_time;
__constant__ uint8_t digest[DIGEST_SIZE];
__constant__ char nusnet_id[NUSNET_ID_SIZE + 1] = NUSNET_ID;
__constant__ uint64_t target; // target value is 64 bit

__device__ void construct_input(uint8_t __restrict__ input[52], uint64_t* __restrict__ nonce)
{
    size_t cur = 0;
    // Fill in the timestamp
    memcpy(input, &cur_time, TIME_SIZE);
    cur += TIME_SIZE;
    // Fill in previous digest
    memcpy((uint8_t*)input + cur, digest, DIGEST_SIZE);
    cur += DIGEST_SIZE;
    // Fill in NUSNET ID
    memcpy((uint8_t*)input + cur, nusnet_id, NUSNET_ID_SIZE);
    cur += NUSNET_ID_SIZE;
    // Fill in nonce
    memcpy((uint8_t*)input + cur, nonce, NONCE_SIZE);
}

__global__ void find_hash()
{
    int prev_is_found;
    uint8_t input[52], hash[32];
    uint64_t to_compare;
    uint64_t nonce = offset + blockIdx.x * NUM_THREADS + threadIdx.x;
    if (is_found) {
        return;
    }
    construct_input(input, &nonce);
    sha256(hash, input, 52);
    memcpy(&to_compare, hash, sizeof(uint64_t));
    if (to_compare < target) {
        // Test-and-set to prevent race condition of multiple writes
        prev_is_found = atomicExch(&is_found, 1);
        if (!prev_is_found) {
            nonce_found = nonce;
            memcpy(result, hash, DIGEST_SIZE);
        }
    }
}

void process_digest(char prev_hash[65], uint8_t digest[DIGEST_SIZE])
{
    int i;
    char cur_byte_string[3];
    // Set last element to null
    cur_byte_string[2] = 0;
    for (i = 0; i < DIGEST_SIZE; i++) {
        cur_byte_string[0] = prev_hash[i * 2];
        cur_byte_string[1] = prev_hash[(i * 2) + 1];
        digest[i] = strtol(cur_byte_string, NULL, 16);
    }
}

void print_digest(uint8_t __restrict__ digest[DIGEST_SIZE])
{
    int i;
    for (i = 0; i < DIGEST_SIZE; i++) {
        printf("%x", digest[i]);
    }
    printf("\n");
}

int main(int argc, char** argv)
{
    char prev_hash[65]; // SHA-256 is 64 chars long
    uint64_t target_local;
    uint32_t cur_time_local = time(NULL);

    scanf("%s", prev_hash);
    scanf("%lu", &target_local);
    process_digest(prev_hash, digest);

    hipMemcpyToSymbol(HIP_SYMBOL(cur_time), &cur_time_local, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(target), &target_local, sizeof(uint64_t));

#ifdef DEBUG
    puts("Input:");
    printf("%s\n", prev_hash);
    printf("%lu\n", target_local);
#endif

    while (!is_found) {
#ifdef DEBUG
        printf("Trying offset %lu\n", offset);
#endif
#ifdef SEQUENTIAL
        find_hash<<<1, 1>>>();
        offset++;
#else
        find_hash<<<NUM_BLOCKS, NUM_THREADS>>>();
        offset += OFFSET_STEP;
#endif
        hipDeviceSynchronize();
    }

    puts(NUSNET_ID);
    printf("%u\n", cur_time_local);
    printf("%lu\n", nonce_found);
    print_digest(result);
    return 0;
}
