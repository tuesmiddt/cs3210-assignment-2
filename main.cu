#include "hip/hip_runtime.h"
#include "hash.h"
#include <stdbool.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>

// #define SEQUENTIAL
#define DEBUG

// Tesla V100: 84 SMs, each with 64 INT32 cores
#define NUM_BLOCKS 84
#define NUM_THREADS 64

__managed__ int result;
__managed__ bool is_found = false;

__global__ void find_hash()
{
    if (is_found) {
        return;
    }
}

int main(int argc, char** argv)
{
    char prev_hash[65]; // SHA-256 is 64 chars long
    uint64_t target; // target value is 64 bit
    scanf("%s", prev_hash);
    scanf("%lu", &target);

#ifdef DEBUG
    puts("Input:");
    printf("%s\n", prev_hash);
    printf("%lu\n", target);
#endif

#ifdef SEQUENTIAL
    find_hash<<<1, 1>>>();
#else
    find_hash<<<NUM_BLOCKS, NUM_THREADS>>>();
#endif
    return 0;
}
